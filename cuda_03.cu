#include "hip/hip_runtime.h"

#include <stdio.h>
#define SCREEN_WIDTH 6
#define SCREEN_HEIGHT 4

__global__ void GPU_kernel(int maxcount)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int actual_index = x + (y * blockDim.x * gridDim.x);
	
	if (actual_index >= maxcount) return;
	printf("Actual pixel: %i , %i\n", x, y);
}

int main(void)
{
	int ThreadsX = 3, ThreadsY = 2;
	dim3 blockarray(( SCREEN_WIDTH + ThreadsX - 1) / ThreadsX, (SCREEN_HEIGHT + ThreadsY - 1) / ThreadsY);
	dim3 threadarray(ThreadsX, ThreadsY);

	GPU_kernel <<< blockarray, threadarray >>> ( SCREEN_WIDTH * SCREEN_HEIGHT);
	hipDeviceSynchronize();
	printf("Execution done!\n");
	return 0;
}
