#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel()
{
	printf("block: %i, thread:%i\n", blockIdx.x, threadIdx.x);
}

int main(void)
{
	GPU_kernel << < 10, 2 >> > ();
	hipDeviceSynchronize();
	printf("Execution done!\n");
	return 0;
}
