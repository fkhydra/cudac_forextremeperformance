#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void set_value(int *c, int number)
{
    c[0] = number;
}

int main()
{
	int *dev0_data;
	int *dev1_data;
	int data=0;

	hipSetDevice(0);
	hipMalloc((void**)&dev0_data, 1 * sizeof(int));
	hipMemcpy(dev0_data, &data, 1 * sizeof(int), hipMemcpyHostToDevice);	
	hipSetDevice(1);
	hipMalloc((void**)&dev1_data, 1 * sizeof(int));	
	hipMemcpy(dev0_data, &data, 1 * sizeof(int), hipMemcpyHostToDevice);
	printf("Memory allocated...\n");

	hipSetDevice(0);
	set_value << <1, 1 >> > (dev0_data,1);
	hipDeviceSynchronize();
	hipSetDevice(1);
	set_value << <1, 1 >> > (dev1_data, 9);
	hipDeviceSynchronize();
	printf("Kernels ok...\n");

	hipSetDevice(0);
	hipMemcpy(&data, dev0_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("DEV0: %i\n", data);
	hipSetDevice(1);
	hipMemcpy(&data, dev1_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("DEV1: %i\n", data);

	hipMemcpyPeer(dev0_data,0, dev1_data,1,1*sizeof(int));
	printf("Swap ok...\n");

	hipSetDevice(0);
	hipMemcpy(&data, dev0_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("DEV0: %i\n",data);
	hipSetDevice(1);
	hipMemcpy(&data, dev1_data, 1 * sizeof(int), hipMemcpyDeviceToHost);
	printf("DEV1: %i\n", data);
	
	hipFree(dev0_data);
	hipFree(dev1_data);
    return 0;
}
