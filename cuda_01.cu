#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void GPU_kernel(int maxcount)
{
	int i;
	int startindex = threadIdx.x + (blockIdx.x * blockDim.x);
	int step = blockDim.x * gridDim.x;
	for (i = startindex; i < maxcount; i += step)
	{
		printf("%i\n", i);
	}
}

int main(void)
{
	int threads = 128;
	int blocks = (1000000 + threads - 1) / threads;

	GPU_kernel << < blocks , threads >> > (1000000);
	hipDeviceSynchronize();
	printf("Execution done!\n");
	return 0;
}
