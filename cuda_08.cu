#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <windows.h>
#include <d2d1.h>
#include <d2d1helper.h>
#pragma comment(lib, "d2d1")

//*****double buffering*****
#define SCREEN_WIDTH 1920
#define SCREEN_HEIGHT 1000

D2D1_RECT_U display_area;
ID2D1Bitmap *image_container = NULL;
unsigned int *dev_image_data, image_data[SCREEN_WIDTH * SCREEN_HEIGHT];
typedef struct Vec3f {
	float x, y, z;
};
//**************************************

ID2D1Factory* pD2DFactory = NULL;
ID2D1HwndRenderTarget* pRT = NULL;

#define HIBA_00 TEXT("Error:Program initialisation process.")
HINSTANCE hInstGlob;
int SajatiCmdShow;
char szClassName[] = "WindowsApp";
HWND Form1; //Windows handler

LRESULT CALLBACK WndProc0(HWND, UINT, WPARAM, LPARAM);
void D2D_drawing(ID2D1HwndRenderTarget* pRT);

//*****double buffering*****
void create_main_buffer(void);
void CUDA_cleanup_main_buffer(void);
void swap_main_buffer(void);
//**************************************

//*****Drawing algorithms*****
__device__ void CUDA_SetPixel(int x1, int y1, int color, unsigned int *puffer);
__device__ void CUDA_DrawLine(int x1, int y1, int x2, int y2, int color, unsigned int *puffer);
__device__ void CUDA_FillTriangle(int x1, int y1, int x2, int y2, int x3, int y3, int color, unsigned int *puffer);
//**************************************

//*********************************
//The main entry point of our program
//*********************************
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, PSTR szCmdLine, int iCmdShow)
{
	static TCHAR szAppName[] = TEXT("StdWinClassName");
	HWND hwnd;
	MSG msg;
	WNDCLASS wndclass0;
	SajatiCmdShow = iCmdShow;
	hInstGlob = hInstance;

	//*********************************
	//Preparing Windows class
	//*********************************
	wndclass0.style = CS_HREDRAW | CS_VREDRAW;
	wndclass0.lpfnWndProc = WndProc0;
	wndclass0.cbClsExtra = 0;
	wndclass0.cbWndExtra = 0;
	wndclass0.hInstance = hInstance;
	wndclass0.hIcon = LoadIcon(NULL, IDI_APPLICATION);
	wndclass0.hCursor = LoadCursor(NULL, IDC_ARROW);
	wndclass0.hbrBackground = (HBRUSH)GetStockObject(LTGRAY_BRUSH);
	wndclass0.lpszMenuName = NULL;
	wndclass0.lpszClassName = TEXT("WIN0");

	//*********************************
	//Registering our windows class
	//*********************************
	if (!RegisterClass(&wndclass0))
	{
		MessageBox(NULL, HIBA_00, TEXT("Program Start"), MB_ICONERROR);
		return 0;
	}

	//*********************************
	//Creating the window
	//*********************************
	Form1 = CreateWindow(TEXT("WIN0"),
		TEXT("CUDA - DIRECT2D"),
		(WS_OVERLAPPED | WS_SYSMENU | WS_THICKFRAME | WS_MAXIMIZEBOX | WS_MINIMIZEBOX),
		50,
		50,
		SCREEN_WIDTH,
		SCREEN_HEIGHT,
		NULL,
		NULL,
		hInstance,
		NULL);

	//*********************************
	//Displaying the window
	//*********************************
	ShowWindow(Form1, SajatiCmdShow);
	UpdateWindow(Form1);

	//*********************************
	//Activating the message processing for our window
	//*********************************
	while (GetMessage(&msg, NULL, 0, 0))
	{
		TranslateMessage(&msg);
		DispatchMessage(&msg);
	}
	return msg.wParam;
}

//*********************************
//The window's callback funtcion: handling events
//*********************************
LRESULT CALLBACK WndProc0(HWND hwnd, UINT message, WPARAM wParam, LPARAM lParam)
{
	HDC hdc;
	PAINTSTRUCT ps;

	switch (message)
	{
	//*********************************
	//When creating the window
	//*********************************
	case WM_CREATE:
		D2D1CreateFactory(D2D1_FACTORY_TYPE_SINGLE_THREADED, &pD2DFactory);
		pD2DFactory->CreateHwndRenderTarget(
			D2D1::RenderTargetProperties(),
			D2D1::HwndRenderTargetProperties(
				hwnd, D2D1::SizeU(SCREEN_WIDTH, SCREEN_HEIGHT)),
			&pRT);
		create_main_buffer();
		hipMalloc((void**)&dev_image_data, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(unsigned int));
		return 0;
	//*********************************
	//to eliminate color flickering
	//*********************************
	case WM_ERASEBKGND:
		return (LRESULT)1;
	//*********************************
	//Repainting the client area of the window
	//*********************************
	case WM_PAINT:
		hdc = BeginPaint(hwnd, &ps);
		EndPaint(hwnd, &ps);
		D2D_drawing(pRT);
		return 0;
	//*********************************
	//Closing the window, freeing resources
	//*********************************
	case WM_CLOSE:
		pRT->Release();
		pD2DFactory->Release();
		hipFree(dev_image_data);
		DestroyWindow(hwnd);
		return 0;
	//*********************************
	//Destroying the window
	//*********************************
	case WM_DESTROY:
		PostQuitMessage(0);
		return 0;
	}
	return DefWindowProc(hwnd, message, wParam, lParam);
}

void D2D_drawing(ID2D1HwndRenderTarget* pRT)
{
	CUDA_cleanup_main_buffer();
	//render_objects<<<blocks,threads >>>(dev_image_data);
	hipDeviceSynchronize();
	hipMemcpy(image_data, dev_image_data, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(unsigned int), hipMemcpyDeviceToHost);
	swap_main_buffer();
}

void create_main_buffer(void)
{
	pRT->CreateBitmap(D2D1::SizeU(SCREEN_WIDTH, SCREEN_HEIGHT),
		D2D1::BitmapProperties(D2D1::PixelFormat(DXGI_FORMAT_B8G8R8A8_UNORM,
			D2D1_ALPHA_MODE_IGNORE)), &image_container);
}

void CUDA_cleanup_main_buffer(void)
{
	hipMemset(dev_image_data, 255, SCREEN_HEIGHT*SCREEN_WIDTH * sizeof(unsigned int));
}

void swap_main_buffer(void)
{
	display_area.left = 0;
	display_area.top = 0;
	display_area.right = SCREEN_WIDTH;
	display_area.bottom = SCREEN_HEIGHT;
	image_container->CopyFromMemory(&display_area, image_data, SCREEN_WIDTH * sizeof(unsigned int));
	pRT->BeginDraw();
	pRT->DrawBitmap(image_container, D2D1::RectF(0.0f, 0.0f, SCREEN_WIDTH, SCREEN_HEIGHT), 1.0f, D2D1_BITMAP_INTERPOLATION_MODE_NEAREST_NEIGHBOR, NULL);
	pRT->EndDraw();
}

__device__ void CUDA_SetPixel(int x1, int y1, int color, unsigned int *puffer)
{
	puffer[(y1 * SCREEN_WIDTH) + x1] = color;
}

__device__ void CUDA_DrawLine(int x1, int y1, int x2, int y2, int color, unsigned int *puffer)
{
	bool flip = false;
	int swap, offset;

	if (abs(x2 - x1) < 2 && abs(y2 - y1) < 2)
	{
		puffer[(y2*SCREEN_WIDTH) + x2] = color; return;
	}
	if (abs(x1 - x2) < abs(y1 - y2))
	{
		swap = x1;
		x1 = y1;
		y1 = swap;

		swap = x2;
		x2 = y2;
		y2 = swap;
		flip = true;
	}
	if (x1 > x2)
	{
		swap = x1;
		x1 = x2;
		x2 = swap;

		swap = y1;
		y1 = y2;
		y2 = swap;
	}
	int dx = x2 - x1;
	int dy = y2 - y1;

	int marker1 = abs(dy) * 2;
	int marker2 = 0;
	int y = y1, x;

	if (flip)
	{
		for (x = x1; x <= x2; ++x)
		{
			offset = (x * SCREEN_WIDTH);
			puffer[offset + y] = color;
			marker2 += marker1;
			if (marker2 > dx)
			{
				y += (y2 > y1 ? 1 : -1);
				marker2 -= dx * 2;
			}
		}
	}
	else
	{
		for (x = x1; x <= x2; ++x)
		{
			offset = (y * SCREEN_WIDTH);
			puffer[offset + x] = color;
			marker2 += marker1;
			if (marker2 > dx)
			{
				y += (y2 > y1 ? 1 : -1);
				marker2 -= dx * 2;
			}
		}
	}
}

void CUDA_FillTriangle(int x1, int y1, int x2, int y2, int x3, int y3, int color, unsigned int *puffer)
{
	int Ax, Ay, Bx, By, i, j;
	int swapx, swapy, offset, maxoffset = SCREEN_HEIGHT * SCREEN_WIDTH;
	if (y1 == y2 && y1 == y3) return;

	if (y1 > y2)
	{
		swapx = x1;
		swapy = y1;
		x1 = x2;
		y1 = y2;
		x2 = swapx;
		y2 = swapy;
	}
	if (y1 > y3)
	{
		swapx = x1;
		swapy = y1;
		x1 = x3;
		y1 = y3;
		x3 = swapx;
		y3 = swapy;
	}
	if (y2 > y3)
	{
		swapx = x3;
		swapy = y3;
		x3 = x2;
		y3 = y2;
		x2 = swapx;
		y2 = swapy;
	}
	int t_height = y3 - y1;
	for (i = 0; i < t_height; ++i)
	{
		bool lower_part = i > y2 - y1 || y2 == y1;
		int part_height = lower_part ? y3 - y2 : y2 - y1;
		float alpha = (float)i / t_height;
		float beta = (float)(i - (lower_part ? y2 - y1 : 0)) / part_height;
		Ax = x1 + (x3 - x1)*alpha;
		Ay = y1 + (y3 - y1)*alpha;
		Bx = lower_part ? x2 + (x3 - x2)*beta : x1 + (x2 - x1)*beta;
		By = lower_part ? y2 + (y3 - y2)*beta : y1 + (y2 - y1)*beta;
		if (Ax > Bx)
		{
			swapx = Ax;
			swapy = Ay;
			Ax = Bx;
			Ay = By;
			Bx = swapx;
			By = swapy;
		}

		offset = (y1 + i)*SCREEN_WIDTH;
		for (j = Ax; j < Bx; ++j)
		{
			if (offset + j > maxoffset) continue;
			puffer[offset + j] = color;
		}
	}
}
